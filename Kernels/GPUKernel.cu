#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdio>
#include <fstream>
#include <string>
#include <iostream>


#define WIDTH 32 // same as num of thread

using namespace std;

// device code - kernel code
__global__ void greyscaleToLbpConversion(unsigned char* in, unsigned char* out, int w, int h)
{


	// Calculate global row and column indices based on the block and thread indices
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row = by * WIDTH + ty;
	int col = bx * WIDTH + tx;

	// decleration for LBP vars
	int LBP[8];
	int LBPValue = 0;
	int lbp_i = 0;


	if (row < h && col < w ) {
			unsigned char center_pixel = in[row * w + col]; // 1D access for 2D array

			for (int i = -1; i <= 1; i++) {
				for (int j = -1; j <= 1; j++) {
					if (row + i >= 0 && row + i < h && col + j >= 0 && col + j < w) {
						// check if nighbor pixel is center 
						if ((i == 0 && j == 0))
							continue;
				
						// check neighbors with center
						if (in[(i + row) * w + (j + col)] >= center_pixel) 
							LBP[lbp_i] = 1; // neighbor > center -> LBP = 1
						else if (in[(i + row) * w + (j + col)] <= center_pixel) 
							LBP[lbp_i] = 0; // neighbor < center -> LBP = 0
						
						// increment LBP index
						lbp_i++;
					}// extra checking of boundry 
			}// inner col loop
		}// outer row loop
	}// if boundry

	// convert LBP value to binary
	LBPValue = LBP[0] * (128) + LBP[1] * (64) + LBP[2] * (32)
		+ LBP[3] * (16) + LBP[4] * (8) + LBP[5] * (4) + LBP[6] * (2)
		+ LBP[7] * (1);

	// store the decimal LBP value in output array
	out[row * w + col] = LBPValue; 
}// device code - kernel code


bool readPGM(const std::string& filename, unsigned char** image, int* height, int* width) {
	std::ifstream file(filename, std::ios::binary);
	if (!file.is_open()) {
		std::cerr << "Error opening file!" << std::endl;
		return false;
	}

	std::string magic;
	std::getline(file, magic);  // Read the magic number (P5)
	if (magic != "P5") {
		std::cerr << "Not a valid P5 PGM file!" << std::endl;
		return false;
	}

	file >> *width >> *height;
	int maxVal;
	file >> maxVal;
	file.ignore();

	*image = new unsigned char[*width * *height];
	file.read(reinterpret_cast<char*>(*image), *width * *height);
	file.close();
	return true;
}

void writePGM(const std::string& filename, unsigned char* image, int height, int width) {
	std::ofstream file(filename, std::ios::binary);
	if (!file.is_open()) {
		std::cerr << "Error opening file!" << std::endl;
		return;
	}

	file << "P5\n" << width << " " << height << "\n255\n";
	file.write(reinterpret_cast<char*>(image), width * height);
	file.close();
}


int main()
{
	std::string inputPath = "C:/Users/User/OneDrive/Desktop/archProject/image128.pgm";  // P5 image file path after greyscale
	std::string outputPath = "C:/Users/User/OneDrive/Desktop/archProject/outputGPU-128.pgm";  // output file path

	unsigned char* pic_in = nullptr; // pointer in host (in is gray)
	unsigned char* pic_out = nullptr; // pointer in host (out is LBP)
	unsigned char* d_pic_in = nullptr; // pointer of device (in is gray)
	unsigned char* d_pic_out = nullptr; // pointer of device (out is LBP)
	int width, height;
	int grey_size; // pixels in gray scale is 1 element

	// Read PGM file
	if (!readPGM(inputPath, &pic_in, &height, &width)) {
		return 1;
	}

	grey_size = width * height * sizeof(unsigned char);

	// allocate memory for device arrays
	hipMalloc((void**)&d_pic_in, grey_size);
	hipMalloc((void**)&d_pic_out, grey_size);

	// creating CUDA events (timer to see how much time from gray to LBP)
	hipEvent_t start, stop;
	hipEventCreate(&start); // create (& is address)
	hipEventCreate(&stop);

	// copy host array to device (gray image)
	hipMemcpy(d_pic_in, pic_in, grey_size, hipMemcpyHostToDevice);

	// Initialize thread block and kernel grid dimensions
	int threads = 32; // given, size of one block
	int blocksX = ceil(width * 1.0 / WIDTH); //threads
	int blocksY = ceil(height * 1.0 / WIDTH); //threads

	dim3 BlockSize(WIDTH, WIDTH, 1); // 2D block
	dim3 GridSize(blocksX, blocksY, 1); // 2D thread

	// place start event into the default stream (timer is started ( to know how to compare CPU VS GPU))
	hipEventRecord(start);

	// kernel call (pointers of device)
	greyscaleToLbpConversion << <GridSize, BlockSize >> > (d_pic_in, d_pic_out, width, height);

	// place stop event into the default stream
	hipEventRecord(stop);

	// block CPU execution until the specified event is recorded
	hipEventSynchronize(stop);

	float milliseconds = 0;
	// returns in the first argument the number of milliseconds time elapsed between the recording start and stop
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("GPU time is %.3f milliseconds\n", milliseconds);

	// allocate host memory for the output image
	pic_out = (unsigned char*)malloc(grey_size);

	// copy result from device to host
	hipMemcpy(pic_out, d_pic_out, grey_size, hipMemcpyDeviceToHost);

	// wait for device to complete all tasks
	hipDeviceSynchronize();

	// Write the output image
	writePGM(outputPath, pic_out, height, width);

	// free
	hipFree(d_pic_in);
	hipFree(d_pic_out);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	free(pic_in);
	free(pic_out);

	std::cout << "LBP processing done on GPU. Output saved to " << outputPath << "\n";

	return 0;
}