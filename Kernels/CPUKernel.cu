#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <bitset>
#include <fstream>  // For std::ofstream
#include <string>   // For std::getline
#include <ctime>    // For clock()

void cpuLBP(unsigned char* Arr, unsigned char* ArrOut, int height, int width) {
	// CPU implementation for the LBP kernel ;D
	// we will skip the boundry pixels becuase they dont have 8 neighbors, start with index 1
    for (int i = 1; i < height - 1; i++) //iterate through the rows
    {
        for (int j = 1; j < width - 1; j++) //iterate throught the columns
        {
            unsigned char center_pixel = Arr[i * width + j]; // 1D access for 2D array
			int lbp = 0; // initialize LBP value
            
            // check the neighbors
            
            /*
            B0  B1  B2
            B7  C   B3
            B6  B5  B4
            */

            if (Arr[(i - 1) * width + (j - 1)] >= center_pixel)
                lbp += 1; //B0
            if (Arr[(i - 1) * width + j] >= center_pixel)
                lbp += 2; //B1
			if (Arr[(i - 1) * width + (j + 1)] >= center_pixel)
				lbp += 4; //B2
			if (Arr[i * width + (j + 1)] >= center_pixel)
				lbp += 8; //B3
			if (Arr[(i + 1) * width + (j + 1)] >= center_pixel)
				lbp += 16; //B4
			if (Arr[(i + 1) * width + j] >= center_pixel)
				lbp += 32; //B5
			if (Arr[(i + 1) * width + (j - 1)] >= center_pixel)
				lbp += 64; //B6
			if (Arr[i * width + (j - 1)] >= center_pixel)
				lbp += 128; //B7

			// convert the LBP value to binary (8-bits) byte
            unsigned char binary_lbp = static_cast<unsigned char>(lbp);

			ArrOut[i * width + j] = lbp; // store the decimal LBP value in the output array

        }
    }

}


// Function to read PGM (P5 binary) image
bool readPGM(const std::string& filename, unsigned char** image, int* height, int* width) {
    std::ifstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error opening file!" << std::endl;
        return false;
    }

    std::string magic;
    std::getline(file, magic);  // Read the magic number (P5)
    if (magic != "P5") {
        std::cerr << "Not a valid P5 PGM file!" << std::endl;
        return false;
    }

    file >> *width >> *height;  // Read width and height
    int maxVal;
    file >> maxVal;  // Maximum pixel value, typically 255
    file.ignore();  // Ignore the newline after the header

    *image = new unsigned char[*width * *height];  // Allocate memory for image data
    file.read(reinterpret_cast<char*>(*image), *width * *height);  // Read pixel data
    file.close();
    return true;
}

// Function to write output PGM (binary) image
void writePGM(const std::string& filename, unsigned char* image, int height, int width) {
    std::ofstream file(filename, std::ios::binary);
    if (!file.is_open()) {
        std::cerr << "Error opening file!" << std::endl;
        return;
    }

    file << "P5\n";  // Magic number for PGM
    file << width << " " << height << "\n";
    file << "255\n";  // Max color value
    file.write(reinterpret_cast<char*>(image), width * height);  // Write pixel data
    file.close();
}


int main() {
    std::string filePath = "C:/Users/User/OneDrive/Desktop/archProject/image1024.pgm";  // P5 image file path after greyscale
    std::string outputPath = "C:/Users/User/OneDrive/Desktop/archProject/output1024.pgm";  // output file path

    // Read PGM file
    unsigned char* image = nullptr;
    int width, height;
    if (!readPGM(filePath, &image, &height, &width)) {
        return 1;
    }

    // Allocate memory for the output image
    unsigned char* outImage = (unsigned char*)malloc(width * height);

    // Measure CPU time for LBP processing
    clock_t start = clock();  // Start measuring time

    // Apply LBP processing
    cpuLBP(image, outImage, height, width);

    clock_t end = clock();    // End measuring time
    double duration = double(end - start) * 1000.0 / CLOCKS_PER_SEC;  // Calculate the duration in seconds
    std::cout << "CPU LBP processing time: " << duration << " ms\n";

    // Write the output image
    writePGM(outputPath, outImage, height, width);

    // Free memory
    free(image);
    free(outImage);

    std::cout << "LBP processing done. Output saved to " << outputPath << "\n";
    return 0;
}

